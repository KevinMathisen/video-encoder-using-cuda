#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <limits.h>
#include "me.h"
#include "c63.h"

/**
 * Function for calculating sum of absolute difference between 
 * a block in orgin frame and a block in ref frame
 * 
 * @param orig      Pointer to top left corner of 8x8 block in orig
 * @param ref       Pointer to top left corner of 8x8 block in ref
 * @param stride    Width of frame, used for indexing
 * 
 * @return          Sum of absolute difference between blocks
 */
__device__ __forceinline__ int sad_block_8x8_device(const uint8_t share_orig[8][8], 
    const uint8_t share_ref[32][32], int ref_x, int ref_y)
{
    int u, v;
    int result = 0;

    for (v = 0; v < 8; ++v)
    {
        for (u = 0; u < 8; u+=4)
        {
            // Load 4 bytes at a time for memory coalescing
            result += abs(share_ref[ref_y + v][ref_x+u] - share_orig[v][u]);
            result += abs(share_ref[ref_y + v][ref_x+u+1] - share_orig[v][u+1]);
            result += abs(share_ref[ref_y + v][ref_x+u+2] - share_orig[v][u+2]);
            result += abs(share_ref[ref_y + v][ref_x+u+3] - share_orig[v][u+3]);
        }
    }
    return result;
}

/**
 * Uses warp level primitive to find the smallest sad value and its offset for a warp
 * 
 * @param sad   SAD value of a thread/candidate 
 * @param mv_x  X motion vector offset for candidate
 * @param mv_y  Y motion vector offset for candidate
 */
__device__ __forceinline__ void warp_min_reduction(int &sad, int &mv_x, int &mv_y)
{
    #pragma unroll
    for (int offset = 16; offset > 0; offset /= 2)
    {
        // Get the value to compare with
        int sad_compare = __shfl_xor_sync(0xFFFFFFFF, sad, offset);   // (assume 32 lanes in each warp because we 
        int mv_x_compare = __shfl_xor_sync(0xFFFFFFFF, mv_x, offset); //  have search range 16 so 1025 threads.
        int mv_y_compare = __shfl_xor_sync(0xFFFFFFFF, mv_y, offset); //  could use __activemask() instead of 0xFFFFFFFF)

        if (sad_compare < sad) { // Update values to the one with smallest sad
            sad = sad_compare;
            mv_x = mv_x_compare;
            mv_y = mv_y_compare;
        }
    }
}

/**
 * Kernel for doing motion estimation on a given macroblock, and finding the
 * offset with the smallest sad to use in the encoding. 
 * 
 * @param d_orig    Frame we are encoding
 * @param d_ref     Frame we are using as reference for finding residuals
 * @param d_mbs     Where we store offset for each macroblock
 * @param range     Search range, i.e. how much to search in reference. Is halved for u and v
 * @param w         width of frame
 * @param h         height of frame
 * @param mb_cols   Number of columns
 * @param mb_rows   Number of rows
 */
__global__ void me_kernel(const uint8_t *d_orig, uint8_t *d_ref,
struct macroblock *d_mbs, int range, int w, int h, int mb_cols, int mb_rows)
{
    // Macroblock index from the grid
    int mb_x = blockIdx.x, mb_y = blockIdx.y;

    // Return if outside of valid blocks
    if (mb_x >= mb_cols || mb_y >= mb_rows) return;

    // Find where orig block starts
    int mx = mb_x * 8, my = mb_y * 8;

    // Allocate shared memory for original 8x8 block and 32x32 reference block
    __shared__ uint8_t share_orig[8][8];
    __shared__ uint8_t share_ref[32][32];

    // Thread index to identify which candidate
    int tid_x = threadIdx.x, tid_y = threadIdx.y;

    // load original 8x8 block into shared memory
    if (tid_x < 8 && tid_y < 8)
        share_orig[tid_y][tid_x] = d_orig[(my+tid_y)*w + (mx+tid_x)];
    
    // Calculate left top corner for search area in reference frame
    int search_left = mb_x*8-range, search_top = mb_y*8-range;

    // Calculate where the thread should then start, 
    // i.e. use the thread index to calculcate where in the search area it is
    int x = search_left + tid_x, y = search_top + tid_y;

    // load 32x32 part of reference frame we use to compare into shared memory
    if (x >= 0 && x < w && y >= 0 && y < h)
        share_ref[tid_y][tid_x] = d_ref[y*w+x];
    else
        share_ref[tid_y][tid_x] = 0; // Set reference outside of frame to 0

    __syncthreads(); // ensure orig and ref is in shared memory before continuing

    int sad_value = INT_MAX;

    // If we are within bounds of reference frame 
    // (Does not support partial frame bounds) 
    if (x >= 0 && x <= w-8 && y >= 0 && y <= h-8) 
    {
        sad_value = sad_block_8x8_device(share_orig, share_ref, tid_x, tid_y);
    }

    // Next we need to find the lowest sad_value and its offset
    // Store (sad, mv_x, mv_y) in shared memory for each thread

    // Shared memory for storing sad, mv_x, and my_y:
    __shared__ int s_sad[1024];
    __shared__ int s_mv_x[1024];
    __shared__ int s_mv_y[1024];

    // Get the thread index used to access shared memory
    int tid = threadIdx.y*blockDim.x + threadIdx.x;

    s_sad[tid] = sad_value;
    s_mv_x[tid] = x-mx;
    s_mv_y[tid] = y-my;

    __syncthreads(); // Ensure all thread values are in shared memory before reduce

    // Use reduction to find the minimum sad
    // Start with half of the blocks, then 1/4, 1/8, etc until we only have one left
    for (int stride = blockDim.x*blockDim.y / 2; stride > 0; stride >>= 1)
    {
        if (tid < stride) // only use threads before stride
        {
            if (s_sad[tid + stride] < s_sad[tid]) // Check if we should move data to left
            {
                s_sad[tid] = s_sad[tid+stride];
                s_mv_x[tid] = s_mv_x[tid+stride];
                s_mv_y[tid] = s_mv_y[tid+stride];
            }
        }
        __syncthreads(); // Ensure all threads have copied if needed
    }

    // thread 0 has the smallest sad, return its offset
    if (tid == 0)
    {
        struct macroblock *mb = &d_mbs[mb_y*mb_cols + mb_x];
        mb->mv_x = s_mv_x[0];
        mb->mv_y = s_mv_y[0];
        mb->use_mv = 1; // always assume MV to be beneficial
    }
}

/**
 * Kernel for doing motion compensation, using the offset found in ME for a block
 * to copy a single pixel in the block from the reference to predicted (output)
 * 
 * @param d_out     Where we will place predicted
 * @param d_ref     Reference we will copy from
 * @param d_mbs     Block offsets
 * @param w         Width of pixels
 * @param h         Height of pixels
 * @param mb_cols   Number of columns 
 * @param mb_rows   Number of rows
 */
__global__ void mc_kernel(uint8_t *d_out, const uint8_t *d_ref,
const struct macroblock *d_mbs, int w, int h, int mb_cols, int mb_rows)
{
    // Macroblock index from the grid
    int mb_x = blockIdx.x, mb_y = blockIdx.y;

    // Return if outside of valid blocks
    if (mb_x >= mb_cols || mb_y >= mb_rows) return;

    // Pixel coordinates in original frame
    int x = mb_x*8 + threadIdx.x, y = mb_y*8 + threadIdx.y;

    // Return if pixel out of bounds
    if (x >= w || y >= h) return;

    // Get macroblock offset
    struct macroblock mb = d_mbs[mb_y*mb_cols + mb_x];

    // check if we should use mv, although redundant
    if (!mb.use_mv) return;

    // Compute pixel coordinates in reference
    int ref_x = x + mb.mv_x, ref_y = y + mb.mv_y;
    // Could check if reference is out of bounds, but should not be possible

    // Copy pixel to predicted frame
    d_out[y*w + x] = d_ref[ref_y*w + ref_x];

}