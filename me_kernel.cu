#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <limits.h>
#include "me.h"
#include "c63.h"

/**
 * Function for calculating sum of absolute difference between 
 * a block in orgin frame and a block in ref frame
 * 
 * @param share_orig    Original block to encode
 * @param share_ref     Reference block to compare and calculcate SAD with
 * @param ref_x         Start of reference block in x
 * @param ref_y         Start of reference block in y
 * 
 * @return          Sum of absolute difference between blocks
 */
__device__ __forceinline__ int sad_block_8x8_device(const uint8_t share_orig[8][8], 
    const uint8_t share_ref[40][40], int ref_x, int ref_y)
{
    int u, v;
    int result = 0;

    for (v = 0; v < 8; ++v)
    {
        for (u = 0; u < 8; u+=4)
        {
            // Load 4 bytes at a time for memory coalescing
            result += abs(share_ref[ref_y + v][ref_x+u] - share_orig[v][u]);
            result += abs(share_ref[ref_y + v][ref_x+u+1] - share_orig[v][u+1]);
            result += abs(share_ref[ref_y + v][ref_x+u+2] - share_orig[v][u+2]);
            result += abs(share_ref[ref_y + v][ref_x+u+3] - share_orig[v][u+3]);
        }
    }
    return result;
}

/**
 * Kernel for doing motion estimation on a given macroblock, and finding the
 * offset with the smallest sad to use in the encoding. 
 * 
 * @param d_orig    Frame we are encoding
 * @param d_ref     Frame we are using as reference for finding residuals
 * @param d_mbs     Where we store offset for each macroblock
 * @param range     Search range, i.e. how much to search in reference. Is halved for u and v
 * @param w         width of frame
 * @param h         height of frame
 * @param mb_cols   Number of columns
 * @param mb_rows   Number of rows
 */
__global__ void me_kernel(const uint8_t *d_orig, uint8_t *d_ref,
struct macroblock *d_mbs, int range, int w, int h, int mb_cols, int mb_rows)
{
    // Macroblock index from the grid
    int mb_x = blockIdx.x, mb_y = blockIdx.y;

    // Return if outside of valid blocks
    if (mb_x >= mb_cols || mb_y >= mb_rows) return;

    // Find where orig block starts
    int mx = mb_x * 8, my = mb_y * 8;

    // Allocate shared memory for original 8x8 block and 40x40 reference block
    //   (40x40 as this covers all pixels when search range is 16 and block size 8x8)
    //   (aka (16x2+8)x(16x2+8), results in 24x24 for search range 8 )
    __shared__ uint8_t share_orig[8][8];
    __shared__ uint8_t share_ref[40][40];

    // Thread index to identify which candidate
    int tid_x = threadIdx.x, tid_y = threadIdx.y;

    // Use 64 threads to load original 8x8 block into shared memory
    if (tid_x < 8 && tid_y < 8)
        share_orig[tid_y][tid_x] = d_orig[(my+tid_y)*w + (mx+tid_x)];
    
    // Calculate left top corner for search area in reference frame
    int search_left = mb_x*8-range, search_top = mb_y*8-range;

    // Calculate where the thread should then start, 
    // i.e. use the thread index to calculcate where in the search area it is
    int x = search_left + tid_x, y = search_top + tid_y;

    // For each thread: load main part of reference frame we use to compare into shared memory
    //  (16 search range -> 32x32, 8 search range -> 16x16)
    if (x >= 0 && x < w && y >= 0 && y < h)
        share_ref[tid_y][tid_x] = d_ref[y*w+x];
    else
        share_ref[tid_y][tid_x] = 0; // Set reference outside of frame to 0

    int main_ref_edge = range*2;

    /* Load "edge" of the reference frame, where the width of it is always 7 (as block size is 8x8)
           Results in filling 40x40 for search range 16, and 24x24 for search range 8 */

    // Load "right" edge
    if (tid_x < 7 && x + main_ref_edge < w) 
        share_ref[tid_y][tid_x + main_ref_edge] = (y >= 0 && y < h) ? d_ref[y*w + (x+main_ref_edge)] : 0;
    // Load "bottom" edge
    if (tid_y < 7 && y + main_ref_edge < h) 
        share_ref[tid_y + main_ref_edge][tid_x] = (x >= 0 && x < w) ? d_ref[(y+main_ref_edge)*w + x] : 0;
    // Load "right-bottom" corner
    if (tid_x < 7 && tid_y < 7 && x + main_ref_edge < w && y + main_ref_edge < h) 
        share_ref[tid_y + main_ref_edge][tid_x + main_ref_edge] = d_ref[(y+main_ref_edge)*w + (x+main_ref_edge)];

    /* Ensure orig and ref is in shared memory before continuing */
    __syncthreads();

    int sad_value = INT_MAX;

    // If we are within bounds of reference frame 
    // (Does not support partial frame bounds) 
    if (x >= 0 && x <= w-8 && y >= 0 && y <= h-8) 
    {
        sad_value = sad_block_8x8_device(share_orig, share_ref, tid_x, tid_y);
    }

    /* Next we need to find the lowest sad_value and its offset 
        Use warp level reduction for this */

    int tid = tid_y * blockDim.x + tid_x;
    int lane = tid%32;      // index of thread in warp
    int warp_id = tid/32;   // index of warp

    // Calculate motion vector offset for thread/candidate
    int mv_x = x-mx, mv_y = y-my;

    /* Find lowest sad for each warp
        Do this by doing reduction with shfl_down_sync to end up with  
        lowest SAD and its offset in lane 0 of each warp */
    for (int offset = 16; offset > 0; offset /= 2) 
    {
        int sad_compare = __shfl_down_sync(0xFFFFFFFF, sad_value, offset);  // (assume 32 lanes in each warp because we 
        int mv_x_compare = __shfl_down_sync(0xFFFFFFFF, mv_x, offset);      //  have search range 16/8 -> 1024/256 threads.
        int mv_y_compare = __shfl_down_sync(0xFFFFFFFF, mv_y, offset);      //  could use __activemask() instead of 0xFFFFFFFF)

        if (lane < offset && sad_compare < sad_value) 
        {
            sad_value = sad_compare;
            mv_x = mv_x_compare;
            mv_y = mv_y_compare;
        }
    }

    /* Ensure all warps are done finding their best SAD */
    __syncthreads();

    /* Now we need to find best SAD from the remaning ones! */

    // Find amount of warps
    int num_warps = (blockDim.x*blockDim.y)/32;

    __shared__ int warp_sad[32];
    __shared__ int warp_mv_x[32];
    __shared__ int warp_mv_y[32];

    if (lane == 0) // use best sad from warp
    {
        warp_sad[warp_id] = sad_value;
        warp_mv_x[warp_id] = mv_x;
        warp_mv_y[warp_id] = mv_y;
    }
        
    /* Ensure all warps have written their minimum */
    __syncthreads();

    /* Final reduction using only first warp */
    if (warp_id == 0) 
    {
        // Each thread/lane in first warp will retreive best SAD from each warp
        sad_value = (lane < num_warps) ? warp_sad[lane] : INT_MAX;
        mv_x = (lane < num_warps) ? warp_mv_x[lane] : 0;
        mv_y = (lane < num_warps) ? warp_mv_y[lane] : 0;

        // Find lowest sad for remaining warp values
        for (int offset = num_warps/2; offset > 0; offset /= 2) 
        {
            int sad_compare = __shfl_down_sync(0xFFFFFFFF, sad_value, offset);  // (assume 32 lanes in each warp because we 
            int mv_x_compare = __shfl_down_sync(0xFFFFFFFF, mv_x, offset);      //  have search range 16/8 -> 1024/256 threads.
            int mv_y_compare = __shfl_down_sync(0xFFFFFFFF, mv_y, offset);      //  could use __activemask() instead of 0xFFFFFFFF)

            if (lane < offset && sad_compare < sad_value) 
            {
                sad_value = sad_compare;
                mv_x = mv_x_compare;
                mv_y = mv_y_compare;
            }
        }
    }

    // Thread 0 has the smallest sad, return its offset
    if (tid == 0)
    {
        struct macroblock *mb = &d_mbs[mb_y*mb_cols + mb_x];
        mb->mv_x = mv_x;
        mb->mv_y = mv_y;
        mb->use_mv = 1; // always assume MV to be beneficial
    }
}

/**
 * Kernel for doing motion compensation, using the offset found in ME for a block
 * to copy a single pixel in the block from the reference to predicted (output)
 * 
 * @param d_out     Where we will place predicted
 * @param d_ref     Reference we will copy from
 * @param d_mbs     Block offsets
 * @param w         Width of pixels
 * @param h         Height of pixels
 * @param mb_cols   Number of columns 
 * @param mb_rows   Number of rows
 */
__global__ void mc_kernel(uint8_t *d_out, const uint8_t *d_ref,
const struct macroblock *d_mbs, int w, int h, int mb_cols, int mb_rows)
{
    // Macroblock index from the grid
    int mb_x = blockIdx.x, mb_y = blockIdx.y;

    // Return if outside of valid blocks
    if (mb_x >= mb_cols || mb_y >= mb_rows) return;

    // Pixel coordinates in original frame
    int x = mb_x*8 + threadIdx.x, y = mb_y*8 + threadIdx.y;

    // Return if pixel out of bounds
    if (x >= w || y >= h) return;

    // Get macroblock offset
    struct macroblock mb = d_mbs[mb_y*mb_cols + mb_x];

    // check if we should use mv, although redundant
    if (!mb.use_mv) return;

    // Compute pixel coordinates in reference
    int ref_x = x + mb.mv_x, ref_y = y + mb.mv_y;
    // Could check if reference is out of bounds, but should not be possible

    // Copy pixel to predicted frame
    d_out[y*w + x] = d_ref[ref_y*w + ref_x];

}