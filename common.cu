#include <assert.h>
#include <errno.h>
#include <getopt.h>
#include <math.h>
#include <stdint.h>
#include <stdio.h>
#include <stdlib.h>
#include <string.h>

#include <hip/hip_runtime.h>

#include "common.h"

#define CUDA_CHECK(call)                                                    \
  do {                                                                      \
    hipError_t err = call;                                                \
    if (err != hipSuccess) {                                              \
      fprintf(stderr, "CUDA error %s:%d: %s\n",                             \
              __FILE__, __LINE__, hipGetErrorString(err));                \
      exit(1);                                                             \
    }                                                                      \
  } while (0)

void destroy_frame(struct frame *f)
{
  /* First frame doesn't have a reconstructed frame to destroy */
  if (!f) { return; }

  // Free pinned memory
  CUDA_CHECK(hipHostFree(f->recons->Y));
  CUDA_CHECK(hipHostFree(f->recons->U));
  CUDA_CHECK(hipHostFree(f->recons->V));
  free(f->recons);

  free(f->residuals->Ydct);
  free(f->residuals->Udct);
  free(f->residuals->Vdct);
  free(f->residuals);

  // Free pinned memory
  CUDA_CHECK(hipHostFree(f->predicted->Y));
  CUDA_CHECK(hipHostFree(f->predicted->U));
  CUDA_CHECK(hipHostFree(f->predicted->V));
  free(f->predicted);

  // Free pinned memory
  CUDA_CHECK(hipHostFree(f->mbs[Y_COMPONENT]));
  CUDA_CHECK(hipHostFree(f->mbs[U_COMPONENT]));
  CUDA_CHECK(hipHostFree(f->mbs[V_COMPONENT]));

  free(f);
}

struct frame* create_frame(struct c63_common *cm, yuv_t *image)
{
  frame *f = (frame*)malloc(sizeof(struct frame));

  f->orig = image;

  // Use pinned memory for reconstructed, as this will be used to encode next frame on GPU
  f->recons = (yuv_t*)malloc(sizeof(yuv_t));
  CUDA_CHECK(hipHostAlloc((void**)&(f->recons->Y), cm->ypw * cm->yph * sizeof(uint8_t), hipHostMallocDefault));
  CUDA_CHECK(hipHostAlloc((void**)&(f->recons->U), cm->upw * cm->uph * sizeof(uint8_t), hipHostMallocDefault));
  CUDA_CHECK(hipHostAlloc((void**)&(f->recons->V), cm->vpw * cm->vph * sizeof(uint8_t), hipHostMallocDefault));

  // Use pinned memory for predicted, as this will be written to from the GPU
  f->predicted = (yuv_t*)malloc(sizeof(yuv_t));
  CUDA_CHECK(hipHostAlloc((void**)&(f->predicted->Y), cm->ypw * cm->yph * sizeof(uint8_t), hipHostMallocDefault));
  CUDA_CHECK(hipHostAlloc((void**)&(f->predicted->U), cm->upw * cm->uph * sizeof(uint8_t), hipHostMallocDefault));
  CUDA_CHECK(hipHostAlloc((void**)&(f->predicted->V), cm->vpw * cm->vph * sizeof(uint8_t), hipHostMallocDefault));

  f->residuals = (dct_t*)malloc(sizeof(dct_t));
  f->residuals->Ydct = (int16_t*)calloc(cm->ypw * cm->yph, sizeof(int16_t));
  f->residuals->Udct = (int16_t*)calloc(cm->upw * cm->uph, sizeof(int16_t));
  f->residuals->Vdct = (int16_t*)calloc(cm->vpw * cm->vph, sizeof(int16_t));

  // Use pinned memory for motion vectors, as this will be written to from the GPU
  CUDA_CHECK(hipHostAlloc((void**)&(f->mbs[Y_COMPONENT]), cm->mb_rows * cm->mb_cols * sizeof(struct macroblock), hipHostMallocDefault));
  CUDA_CHECK(hipHostAlloc((void**)&(f->mbs[U_COMPONENT]), cm->mb_rows/2 * cm->mb_cols/2 * sizeof(struct macroblock), hipHostMallocDefault));
  CUDA_CHECK(hipHostAlloc((void**)&(f->mbs[V_COMPONENT]), cm->mb_rows/2 * cm->mb_cols/2 * sizeof(struct macroblock), hipHostMallocDefault));

  return f;
}

void dump_image(yuv_t *image, int w, int h, FILE *fp)
{
  fwrite(image->Y, 1, w*h, fp);
  fwrite(image->U, 1, w*h/4, fp);
  fwrite(image->V, 1, w*h/4, fp);
}
